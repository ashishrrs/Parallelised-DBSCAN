
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<time.h>
#include <chrono>

double ep;
double pts[1000][50];
int clusters[1000][1000];
int minpts, dim, num_pts;
__global__
void initialization(int *a, int *b, int N)
{

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for (int i = idx; i < N; i += stride)
    {
        a[i] = 0;
        b[i] = 0;
    }
}
__global__
void noise(int *a, int N)
{

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for (int i = idx; i < N; i += stride)
    {
        if (a[i] != 1)
            printf("%d ", i + 1);
    }
}
double sqrd_dist(int i, int j)
{
    double sum = 0;
    for (int k = 0; k < dim ; k++)
    {
        sum += pow(pts[i][k] - pts[j][k], 2);
    }
    return sqrt(sum);
}

void dfs(int i, int *siz, int* vis)
{
    vis[i] = 1;
    printf("%d ", i + 1);
    for (int a  = 0; a < siz[i] ; a++)
    {
        if (vis[clusters[i][a]] != 1)
            dfs(clusters[i][a], siz, vis);
    }
}



int main()
{
    //printf("Enter the ep distance:");
    //scanf("%lf", &ep);
    ep = 1;
    if (ep < 0)
    {
        printf("INVALID EPSILON DISTANCE");
        return 0;
    }

    //printf("Enter the minimum points:");
    //scanf("%d", &minpts);
    minpts = 2;
    if (minpts < 1)
    {
        printf("INVALID MIN PTS");
        return 0;
    }
    //printf("Enter the dimesions of the points:");
    //scanf("%d", &dim);
    dim = 3;
    if (dim < 1)
    {
        printf("INVALID DIMENSIONS");
        return 0;
    }
    //printf("Enter the number of points:");
    //scanf("%d", &num_pts);
    num_pts = 53;
    if (num_pts < 1)
    {
        printf("INVALID NUMBER OF PTS");
        return 0;
    }
    //printf("Enter points:");
    for (int i = 0 ; i < num_pts; i++)
    {
        for (int j = 0; j < dim; j++)
        {
            //scanf("%lf", &pts[i][j]);
            pts[i][j] = rand() % 10;
            //printf("%lf ", pts[i][j]);
        }
        //printf("\n");
    }
    int block_thread[9][2] = {{1, 1}, {1, 10}, {1, 20}, {1, 30}, {1, 40},
        {10, 10}, {20, 10}, {1, num_pts}, {num_pts / 8, num_pts}
    };
    for (int thread = 0; thread < 9; thread++)
    {
        int *siz, *vis;
        size_t size = num_pts * sizeof(int);
        hipMallocManaged(&siz, size);
        size = num_pts * sizeof(int);
        hipMallocManaged(&vis, size);
        auto start = std::chrono::high_resolution_clock::now();
        initialization <<< block_thread[thread][0], block_thread[thread][1]>>>(siz, vis, num_pts);
        hipDeviceSynchronize();
        printf("\n");
        for (int i = 0; i < num_pts - 1; i++)
        {
            for (int j = i + 1; j < num_pts; j++)
            {
                if (i == j)
                    continue;
                if (sqrd_dist(i, j) <= ep)
                {
                    clusters[i][siz[i]] = j;
                    clusters[j][siz[j]] = i;
                    siz[i]++;
                    siz[j]++;
                }
            }
        }

        int cnt = 0;
        for (int i = 0; i < num_pts; i++)
        {
            if (vis[i] != 1 && siz[i] >= minpts)
            {
                cnt++;
                printf("cluster %d : ", cnt);
                dfs(i, siz, vis);
                printf("\n");
            }

        }
        printf("NOISE :");
        noise <<< block_thread[thread][0], block_thread[thread][1]>>>(vis , num_pts);
        hipDeviceSynchronize();
        printf("\n");
        auto end = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);
        printf("Exec time : %ld \n", duration.count());
        hipFree(vis);
        hipFree(siz);
    }
    return 0;
}